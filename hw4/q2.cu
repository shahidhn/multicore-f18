
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#define THREADS_PER_BLOCK 1024



__global__ void parta(int *array, int *B, int n){
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < n){
		atomicAdd(&B[array[index] / 100], 1);
	}	

	__syncthreads();

}


__global__ void partb(int *array, int *B, int n){
	__shared__ int localB[10];
	int i;
		
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < n){
		atomicAdd(&localB[array[index] / 100], 1);
	}

	__syncthreads();

	if (threadIdx.x == 0){
		for (i = 0; i < 10; i++){
			atomicAdd(&B[i], localB[i]);
		}
	}

}

__global__ void partc(int *B, int n){
	int d, val;
	
	for (d = 1; d < n; d = d*2){
		if (threadIdx.x >= d)
			val = B[threadIdx.x-d];
		__syncthreads();
		if (threadIdx.x >= d)
			B[threadIdx.x] += val;
		__syncthreads();	
	}	

}


int main(void) {
	int numcomma = 0;
	char c;
	FILE* stream = fopen("inp.txt", "r");
	while(1){
		c = fgetc(stream);
		if (c == EOF)
			break;
		if (c == ',')
			numcomma ++;
	}
	printf("%d\n", numcomma);
	fclose(stream);
	int array[numcomma+1];

	stream = fopen("inp.txt", "r");
	int i;
	for (i = 0; i <= numcomma; i ++){
		fscanf(stream, "%d,", &array[i]);
	}
	printf("%d\n", array[3]);
	fclose(stream);
	int array_len = numcomma + 1;

	int *d_array;
	int B[10];
	int *d_B;    			
	int size = sizeof(array);
	// Allocate space for device copies of array
	hipMalloc((void **)&d_array, size);
	hipMalloc((void **)&d_B, sizeof(int)*10);
	hipMemcpy(d_array, &array, size, hipMemcpyHostToDevice);

	parta<<<(array_len + THREADS_PER_BLOCK - 1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_array, d_B, array_len);
	hipMemcpy(&B, d_B, sizeof(int)*10, hipMemcpyDeviceToHost);
	hipFree(d_B); hipFree(d_array);

	FILE *q2a = fopen("q2a.txt", "w+");
	for (i = 0; i <= 9; i++){
		fprintf(q2a, "%d", B[i]);
		if (i < 9) fprintf(q2a, ", ");
	}
	fclose(q2a);

	
	
	// Q2b
	
	hipMalloc((void **)&d_array, size);
	hipMalloc((void **)&d_B, sizeof(int)*10);
	hipMemcpy(d_array, &array, size, hipMemcpyHostToDevice);

	partb<<<(array_len + THREADS_PER_BLOCK - 1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_array, d_B, array_len);
	hipMemcpy(&B, d_B, sizeof(int)*10, hipMemcpyDeviceToHost);
	hipFree(d_B); hipFree(d_array);
	
	FILE *q2b = fopen("q2b.txt", "w+");
	for (i = 0; i <= 9; i++){
		fprintf(q2b, "%d", B[i]);
		if (i < 9) fprintf(q2b, ", ");
	}
	fclose(q2b);


	// Q2c


	hipMalloc((void **)&d_B, sizeof(int)*10);
	hipMemcpy(d_B, &B, sizeof(int)*10, hipMemcpyHostToDevice);


	partc<<<1,10>>>(d_B, array_len);
	hipMemcpy(&B, d_B, sizeof(int)*10, hipMemcpyDeviceToHost);
	hipFree(d_B);
	
	FILE *q2c = fopen("q2c.txt", "w+");
	for (i = 0; i <= 9; i++){
		fprintf(q2c, "%d", B[i]);
		if (i < 9) fprintf(q2c, ", ");
	}
	fclose(q2c);









}
