
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#define THREADNUM 16



__global__ void parta(int *array, int *B, int n){
	int chunk_size = n/blockDim.x;
	int i;

	if (threadIdx.x < blockDim.x-1){
		for (i = threadIdx.x * chunk_size; i < threadIdx.x * chunk_size + chunk_size; i++){
			//B[array[i] / 100] += 1;
			atomicAdd(&B[array[i] / 100], 1);
		}
	}
	else{
		for (i = threadIdx.x * chunk_size; i < n; i++){
			//B[array[i] / 100] += 1;
			atomicAdd(&B[array[i] / 100], 1);
		}
	}
	__syncthreads();


}



int main(void) {
	int numcomma = 0;
	char c;
	FILE* stream = fopen("inp.txt", "r");
	while(1){
		c = fgetc(stream);
		if (c == EOF)
			break;
		if (c == ',')
			numcomma ++;
	}
	printf("%d\n", numcomma);
	fclose(stream);
	int array[numcomma+1];

	stream = fopen("inp.txt", "r");
	int i;
	for (i = 0; i <= numcomma; i ++){
		fscanf(stream, "%d,", &array[i]);
	}
	printf("%d\n", array[3]);
	fclose(stream);


	int *d_array;
	int B[10];
	int *d_B;    			
	int size = sizeof(array);
	// Allocate space for device copies of array
	hipMalloc((void **)&d_array, size);
	hipMalloc((void **)&d_B, sizeof(int)*10);
	hipMemcpy(d_array, &array, size, hipMemcpyHostToDevice);

	parta<<<1,THREADNUM>>>(d_array, d_B, numcomma+1);
	hipMemcpy(&B, d_B, sizeof(int)*10, hipMemcpyDeviceToHost);
	hipFree(d_B); hipFree(d_array);
	printf("b0: %d\n", B[0]);
	printf("b1: %d\n", B[1]);
	printf("b5: %d\n", B[5]);
	printf("total count: %d\n", B[0]+B[1]+B[2]+B[3]+B[4]+B[5]+B[6]+B[7]+B[8]+B[9]);
}
