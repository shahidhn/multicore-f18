
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#define THREADS_PER_BLOCK 1024



__global__ void parta(int *array, int *B, int n){
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < n){
		atomicAdd(&B[array[index] / 100], 1);
	}	

	__syncthreads();

}


__global__ void partb(int *array, int *B, int n){
	__shared__ int localB[10];
	int i;
		
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < n){
		atomicAdd(&localB[array[index] / 100], 1);
	}

	__syncthreads();

	if (threadIdx.x == 0){
		for (i = 0; i < 10; i++){
			atomicAdd(&B[i], localB[i]);
		}
	}

}


int main(void) {
	int numcomma = 0;
	char c;
	FILE* stream = fopen("inp.txt", "r");
	while(1){
		c = fgetc(stream);
		if (c == EOF)
			break;
		if (c == ',')
			numcomma ++;
	}
	printf("%d\n", numcomma);
	fclose(stream);
	int array[numcomma+1];

	stream = fopen("inp.txt", "r");
	int i;
	for (i = 0; i <= numcomma; i ++){
		fscanf(stream, "%d,", &array[i]);
	}
	printf("%d\n", array[3]);
	fclose(stream);
	int array_len = numcomma + 1;

	int *d_array;
	int B[10];
	int *d_B;    			
	int size = sizeof(array);
	// Allocate space for device copies of array
	hipMalloc((void **)&d_array, size);
	hipMalloc((void **)&d_B, sizeof(int)*10);
	hipMemcpy(d_array, &array, size, hipMemcpyHostToDevice);

	parta<<<(array_len + THREADS_PER_BLOCK - 1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_array, d_B, array_len);
	hipMemcpy(&B, d_B, sizeof(int)*10, hipMemcpyDeviceToHost);
	hipFree(d_B); hipFree(d_array);
	printf("b0: %d\n", B[0]);
	printf("b1: %d\n", B[1]);
	printf("b5: %d\n", B[5]);
	printf("total count: %d\n", B[0]+B[1]+B[2]+B[3]+B[4]+B[5]+B[6]+B[7]+B[8]+B[9]);
	printf("now correct");
		
	// Q2b
	
	hipMalloc((void **)&d_array, size);
	hipMalloc((void **)&d_B, sizeof(int)*10);
	hipMemcpy(d_array, &array, size, hipMemcpyHostToDevice);

	partb<<<(array_len + THREADS_PER_BLOCK - 1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_array, d_B, array_len);
	hipMemcpy(&B, d_B, sizeof(int)*10, hipMemcpyDeviceToHost);
	hipFree(d_B); hipFree(d_array);
	printf("partB");
	printf("b0: %d\n", B[0]);
	printf("b1: %d\n", B[1]);
	printf("b5: %d\n", B[5]);
	printf("total count: %d\n", B[0]+B[1]+B[2]+B[3]+B[4]+B[5]+B[6]+B[7]+B[8]+B[9]);
	







}
