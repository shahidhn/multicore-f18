/*
Please use "inp.txt" as input file and output/write your results of each question to a separate file named as "q1a.txt", "q1b.txt" etc. The output file should have the same format as the input file. 
You only need to submit three source code files, e.g. q1.cu, q2.cu and q3.cu and the input file "inp.txt". Don't submit any other files.
*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#define NUMBLOCKS 10

__global__ void convertTo1and0(int *array, int *b){
	b[blockIdx.x] = array[blockIdx.x] % 2;
}

__global__ void PPRead(int * b, int * c, int d){
	if (blockIdx.x >= d) c[blockIdx.x] = b[blockIdx.x - d];
}


__global__ void PPWrite(int * b, int * c, int d){
	if (blockIdx.x >= d) b[blockIdx.x] += c[blockIdx.x];
}

__global__ void parallelPrefix(int * b, int *c, int n){
	for (int d = 1; d < n; d *= 2){
//		PPRead<<<n, 1>>>(b, c, d);
//		PPRead<<<n, 1>>>(b, c, d);
	}	
}


int main(void) {
	int numcomma = 0;
	char c;
	FILE* stream = fopen("inp.txt", "r");
	while(1){
		c = fgetc(stream);
		if (c == EOF)
			break;
		if (c == ',')
			numcomma ++;
	}
	fclose(stream);

	int array[numcomma+1];

	stream = fopen("inp.txt", "r");
	int i;
	for (i = 0; i <= numcomma; i ++){
		fscanf(stream, "%d,", &array[i]);
	}
	fclose(stream);


	int *d_array;
	int size = sizeof(array);
	int *d_b;
	int *d_c;
	int *d = (int *) malloc(size);
	int *d_d;

	int *b = (int *) malloc(size);

	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);
	hipMalloc((void **)&d_array, size);
	hipMalloc((void **)&d_d, size);
	hipMemcpy(d_array, &array, size, hipMemcpyHostToDevice);

	convertTo1and0<<<(numcomma+1), 1>>>(d_array, d_b);
	
	for (int d = 1; d <= numcomma; d *= 2){
		PPRead<<<(numcomma+1), 1>>>(d_b, d_c, d);
		PPWrite<<<(numcomma+1), 1>>>(d_b, d_c, d);
	}	
//	parallelPrefix<<<(numcomma+1), 1>>>(d_b, d_c);		

	hipMemcpy(b, d_b, size, hipMemcpyDeviceToHost);
	for (i = 0; i <= numcomma; i ++) printf("%d ", b[i]);

	hipFree(d_array); hipFree(d_d);

	FILE *q3 = fopen("q3.txt", "w+");
	
	free(d);
}




