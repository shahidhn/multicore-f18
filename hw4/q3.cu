/*
Please use "inp.txt" as input file and output/write your results of each question to a separate file named as "q1a.txt", "q1b.txt" etc. The output file should have the same format as the input file. 
You only need to submit three source code files, e.g. q1.cu, q2.cu and q3.cu and the input file "inp.txt". Don't submit any other files.
*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#define NUMBLOCKS 10

__global__ void convertTo1and0(int *array, int *b, int * idx){
	b[blockIdx.x] = array[blockIdx.x] % 2;
	idx[blockIdx.x] = array[blockIdx.x] % 2;
}

__global__ void PPRead(int * b, int * c, int d){
	if (blockIdx.x >= d) c[blockIdx.x] = b[blockIdx.x - d];
}


__global__ void PPWrite(int * b, int * c, int d){
	if (blockIdx.x >= d) b[blockIdx.x] += c[blockIdx.x];
}

__global__ void findOdds(int * array, int * idx, int * b, int * d){
	if (idx[blockIdx.x]){
		d[b[blockIdx.x] - 1] = array[blockIdx.x];
	}	
}

int main(void) {
	int numcomma = 0;
	char c;
	FILE* stream = fopen("inp.txt", "r");
	while(1){
		c = fgetc(stream);
		if (c == EOF)
			break;
		if (c == ',')
			numcomma ++;
	}
	fclose(stream);

	int array[numcomma+1];

	stream = fopen("inp.txt", "r");
	int i;
	for (i = 0; i <= numcomma; i ++){
		fscanf(stream, "%d,", &array[i]);
	}
	fclose(stream);


	int *d_array; // Holds device copy of array
	int size = sizeof(array);
	int *d_b; // Holds parallel prefix sum of d_idx
	int *d_c;
	int *d_idx; // 1 for odd number in array, 0 for even number in array
	int *d = (int *) malloc(size);
	int *d_d; // Holds final answer

	int *b = (int *) malloc(size);

	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);
	hipMalloc((void **)&d_array, size);
	hipMalloc((void **)&d_d, size);
	hipMalloc((void **)&d_idx, size);
	hipMemcpy(d_array, &array, size, hipMemcpyHostToDevice);

	convertTo1and0<<<(numcomma+1), 1>>>(d_array, d_b, d_idx);
	
	for (int d = 1; d <= numcomma; d *= 2){
		PPRead<<<(numcomma+1), 1>>>(d_b, d_c, d);
		PPWrite<<<(numcomma+1), 1>>>(d_b, d_c, d);
	}	

	hipMemcpy(b, d_b, size, hipMemcpyDeviceToHost);
	int final_size = b[numcomma];

	findOdds<<<(numcomma+1), 1>>>(d_array, d_idx, d_b, d_d);

	hipMemcpy(d, d_d, size, hipMemcpyDeviceToHost);
	hipFree(d_array); hipFree(d_d); hipFree(d_b), hipFree(d_c); hipFree(d_idx);

	FILE *q3 = fopen("q3.txt", "w+");
	for (i = 0; i < final_size; i ++){
		fprintf(q3, "%d", d[i]);
		if (i < final_size-1) fprintf(q3, ", ");
	}	

	free(d); free(b);
}




