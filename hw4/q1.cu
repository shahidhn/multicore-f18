/*
Please use "inp.txt" as input file and output/write your results of each question to a separate file named as "q1a.txt", "q1b.txt" etc. The output file should have the same format as the input file. 
You only need to submit three source code files, e.g. q1.cu, q2.cu and q3.cu and the input file "inp.txt". Don't submit any other files.
*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

int main(void) {
	int numcomma = 0;
	char c;
	FILE* stream = fopen("inp.txt", "r");
	while(1){
		c = fgetc(stream);
		if (c == EOF)
			break;
		if (c == ',')
			numcomma ++;
	}
	printf("%d\n", numcomma);
	fclose(stream);
	int array[numcomma+1];

	stream = fopen("inp.txt", "r");
	int i;
	for (i = 0; i <= numcomma; i ++){
		fscanf(stream, "%d,", &array[i]);
	}
	printf("%d\n", array[3]);
	fclose(stream);


}





		// int a[];	 			// host copies of a
		// int *d_a; 	     			// device copies of a
		// int size = sizeof(int);
		
		// // Allocate space for device copies of a, b, c
		// cudaMalloc((void **)&d_a, size);
