/*
Please use "inp.txt" as input file and output/write your results of each question to a separate file named as "q1a.txt", "q1b.txt" etc. The output file should have the same format as the input file. 
You only need to submit three source code files, e.g. q1.cu, q2.cu and q3.cu and the input file "inp.txt". Don't submit any other files.
*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#define THREADNUM 16
#define THREADS_PER_BLOCK 1024

__global__ void min(int *array, int *answer, int n){
	int index = (threadIdx.x + blockIdx.x * blockDim.x)*2;

	int d, val;
	for (d = n; d >= 1;  d = d/2){
		if (index < d){
			val = array[index];
			if (array[index+1] < val)
				val = array[index+1];
		}
		__syncthreads();
		if (index < d){
			array[index/2] = val;
		}
		__syncthreads();
	}
	*answer = array[0];

/*




	int chunk_size = n/blockDim.x;
	int i, localmin = 10000;
	__shared__ int min[THREADNUM];
	if (threadIdx.x < blockDim.x-1){
		for (i = threadIdx.x * chunk_size; i < threadIdx.x * chunk_size + chunk_size; i++){
			if(array[i] < localmin)
				localmin = array[i];
		}
	}
	else{
		for (i = threadIdx.x * chunk_size; i < n; i++){
			if(array[i] < localmin)
				localmin = array[i];
		}
	}
	min[threadIdx.x] = localmin;
	__syncthreads();
	if(threadIdx.x == 0){
		int globalmin = 10000;
		for(i = 0; i < blockDim.x; i ++)
			if(min[i] < globalmin)
				globalmin = min[i];
		*answer = globalmin;
	}
*/
}

__global__ void last_digit(int *array, int *b){
	b[blockIdx.x] = array[blockIdx.x] % 10;
}

int main(void) {
	int numcomma = 0;
	char c;
	FILE* stream = fopen("inp.txt", "r");
	while(1){
		c = fgetc(stream);
		if (c == EOF)
			break;
		if (c == ',')
			numcomma ++;
	}
	fclose(stream);
	int array[numcomma+1];

	stream = fopen("inp.txt", "r");
	int i;
	for (i = 0; i <= numcomma; i ++){
		fscanf(stream, "%d,", &array[i]);
	}
	fclose(stream);


	int *d_array;
	int answer;
	int *d_answer;    			
	int size = sizeof(array);
	int *b = (int *) malloc(size);
	int *d_b;

	// Allocate space for device copies of array
	hipMalloc((void **)&d_array, size);
	hipMalloc((void **)&d_answer, sizeof(int));
	hipMalloc((void **)&d_b, size);

	hipMemcpy(d_array, &array, size, hipMemcpyHostToDevice);

	min<<<(numcomma + THREADS_PER_BLOCK)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_array, d_answer, numcomma+1);
	hipMemcpy(&answer, d_answer, sizeof(int), hipMemcpyDeviceToHost);

	last_digit<<<(numcomma+1), 1>>>(d_array, d_b);
	hipMemcpy(b, d_b, size, hipMemcpyDeviceToHost);

	hipFree(d_answer); hipFree(d_array); hipFree(d_b);

	FILE *q1a = fopen("q1a.txt", "w+");
	fprintf(q1a, "Min: %d\n", answer);
	
	FILE *q1b = fopen("q1b.txt", "w+");
	for (i = 0; i <= numcomma; i ++){
		fprintf(q1b, "%d", b[i]);
		if (i < numcomma) fprintf(q1b, ", ");
	}
	free(b);
}




